#include "hip/hip_runtime.h"
#include "foo.h"

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__global__ void foo(Anchor *pData)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	((Anchor *)(pData + i))->w = i;
	((Anchor *)(pData + i))->h = i * i;

	int test[100]{};
	test[99] = i * i;

    printf("CUDA! %d\n", test[99]);
}

std::vector<Anchor> useCUDA(int size)
{
    Anchor *pData;
	hipMallocManaged(&pData, sizeof(Anchor) * size);
    foo<<<1,size>>>(pData);
    CHECK(hipDeviceSynchronize());

	std::vector<Anchor> result(pData, pData + size);
	hipFree(pData);

	return result;
}

std::vector<Anchor> useCUDA2(int size)
{
    Anchor *gpuData;
	hipMalloc(&gpuData, sizeof(Anchor) * size);
    foo<<<1,size>>>(gpuData);
    CHECK(hipDeviceSynchronize());

	Anchor *cpuData = new Anchor[size];
	hipMemcpyAsync(cpuData, gpuData, sizeof(Anchor)*size, hipMemcpyDeviceToHost);
	hipFree(gpuData);

	std::vector<Anchor> result(cpuData, cpuData + size);
	delete[] cpuData;

	return result;
}


void test()
{
	// H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);

    std::cout << "H now has size " << H.size() << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    // print contents of D
    for (int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;
}